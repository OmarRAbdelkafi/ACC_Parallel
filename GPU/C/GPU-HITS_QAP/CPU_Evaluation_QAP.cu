#ifndef _GPU_EVALUATION_QAP_CU_
#define _GPU_EVALUATION_QAP_CU_

#include <stdint.h>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

typedef int*  type_vector;
typedef int** type_matrix;

/*--------------------------------------------------------------*/
/*       compute the cost difference if elements i and j        */
/*         are transposed in permutation (solution) p           */
/*--------------------------------------------------------------*/
int compute_delta(int n, type_vector a, type_vector b, type_vector p, int i, int j){
  int d; int k;
  d = (a[i*n+i]-a[j*n+j])*(b[p[j]*n+p[j]]-b[p[i]*n+p[i]]) + (a[i*n+j]-a[j*n+i])*(b[p[j]*n+p[i]]-b[p[i]*n+p[j]]);
  for (k = 0; k < n; k = k + 1) if (k!=i && k!=j)
    d = d + (a[k*n+i]-a[k*n+j])*(b[p[k]*n+p[j]]-b[p[k]*n+p[i]]) + (a[i*n+k]-a[j*n+k])*(b[p[j]*n+p[k]]-b[p[i]*n+p[k]]);
  return(d);
 }

/*--------------------------------------------------------------*/
/*      Idem, but the value of delta[i][j] is supposed to       */
/*    be known before the transposition of elements r and s     */
/*--------------------------------------------------------------*/
int compute_delta_part(type_vector a, type_vector b, type_vector p, type_vector delta, int i, int j, int r, int s,int n)
  {
     return ( delta[i*n+j]+(a[r*n+i]-a[r*n+j]+a[s*n+j]-a[s*n+i]) * (b[p[s]*n+p[i]]-b[p[s]*n+p[j]]+b[p[r]*n+p[j]]-b[p[r]*n+p[i]]) +
            (a[i*n+r]-a[j*n+r]+a[j*n+s]-a[i*n+s]) * (b[p[i]*n+p[s]]-b[p[j]*n+p[s]]+b[p[j]*n+p[r]]-b[p[i]*n+p[r]]) );
  }

#endif


