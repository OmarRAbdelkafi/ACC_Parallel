#include "hip/hip_runtime.h"
#ifndef _GPU_EVALUATION_QAP_CU_
#define _GPU_EVALUATION_QAP_CU_

#include <stdint.h>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>


static __global__ void g_compute_delta(int n, int* g_p, int* g_delta, int* c_a, int* c_b){

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int k; int d;
  if(idx < (n*n)){
	  int i=(int)(idx / n);
	  int j=(int)(idx % n);

	  if(i<j){
		    d = (c_a[(i*n)+i]-c_a[(j*n)+j]) * (c_b[(g_p[j]*n)+g_p[j]] - c_b[(g_p[i]*n)+g_p[i]]) +
			(c_a[(i*n)+j] - c_a[(j*n)+i]) * (c_b[(g_p[j]*n)+g_p[i]] - c_b[(g_p[i]*n)+g_p[j]]);

		    for (k = 0; k < n; k = k + 1) 
		    {
				 if (k!=i && k!=j){
				      d = d + (c_a[(k*n)+i]-c_a[(k*n)+j]) * (c_b[(g_p[k]*n)+g_p[j]]-c_b[(g_p[k]*n)+g_p[i]]) +
					 (c_a[(i*n)+k]-c_a[(j*n)+k]) * (c_b[(g_p[j]*n)+g_p[k]]-c_b[(g_p[i]*n)+g_p[k]]);
				 }
		    }
		    g_delta[i*n+j] = d;
	  }
  }
}

static __global__ void g_update_delta(int n, int* g_p, int* g_delta, int* c_a, int* c_b,int i_retained,int j_retained){

  __shared__ int p_j_r;
  __shared__ int p_i_r;

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int k; int d;
  if(idx < (n*n)){
	  int i=(int)(idx / n);
	  int j=(int)(idx % n);

	  p_j_r = g_p[j_retained];
	  p_i_r = g_p[i_retained];
	  __syncthreads();

	  if(i<j){
	       if (i != i_retained && i != j_retained && j != i_retained && j != j_retained){

			 d = g_delta[i*n+j] + (c_a[i_retained*n+i]-c_a[i_retained*n+j]+c_a[j_retained*n+j]-c_a[j_retained*n+i]) *
			 (c_b[p_j_r*n+g_p[i]]-c_b[p_j_r*n+g_p[j]]+c_b[p_i_r*n+g_p[j]]-
			  c_b[p_i_r*n+g_p[i]]) + (c_a[i*n+i_retained]-c_a[j*n+i_retained]+c_a[j*n+j_retained]-
			  c_a[i*n+j_retained]) * (c_b[g_p[i]*n+p_j_r]-c_b[g_p[j]*n+p_j_r]+
			  c_b[g_p[j]*n+p_i_r]-c_b[g_p[i]*n+p_i_r]);

			 __syncthreads();
	       }
	       else{
		 	 d = (c_a[(i*n)+i]-c_a[(j*n)+j]) * (c_b[(g_p[j]*n)+g_p[j]] - c_b[(g_p[i]*n)+g_p[i]]) + (c_a[(i*n)+j] -
			      c_a[(j*n)+i]) * (c_b[(g_p[j]*n)+g_p[i]] - c_b[(g_p[i]*n)+g_p[j]]);

			 for (k = 0; k < n; k = k + 1) 
			 {
				 if (k!=i && k!=j){
					  d = d + (c_a[(k*n)+i]-c_a[(k*n)+j])*(c_b[(g_p[k]*n)+g_p[j]]-c_b[(g_p[k]*n)+g_p[i]]) +
					      (c_a[(i*n)+k]-c_a[(j*n)+k])*(c_b[(g_p[j]*n)+g_p[k]]-c_b[(g_p[i]*n)+g_p[k]]);
				 }
			 }
	       }
	       g_delta[i*n+j] = d;
	  }
  }
}

__host__ void h_compute_delta(int n, int* g_p, int* g_delta, int *c_a, int* c_b) 
{
	//kernel parameters for delta matrix
	//	- block is the size of the thread block
	//	- grid is the number of block computed by the size of the instance
	dim3 Grid((((n*n)/256)+1), 1, 1);
	dim3 Block(256,1,1);
	
	//initialize the matrix of cost of moves
	g_compute_delta<<<Grid, Block>>>(n, g_p, g_delta,c_a,c_b);
	cutilCheckMsg("tabu search: g_compute_delta() execution failed\n");	 
}

__host__ void h_update_delta(int n, int* g_p, int* g_delta, int *c_a, int* c_b, int i_retained, int j_retained) 
{
	//kernel parameters for delta matrix
	//	- block is the size of the thread block
	//	- grid is the number of block computed by the size of the instance
	dim3 Grid((((n*n)/256)+1), 1, 1);
	dim3 Block(256,1,1);
	
	//Update the matrix of move costs
	g_update_delta<<<Grid, Block>>>(n, g_p, g_delta,c_a,c_b,i_retained,j_retained);
	cutilCheckMsg("TS: g_update_delta() execution failed\n");	 
}



#endif


