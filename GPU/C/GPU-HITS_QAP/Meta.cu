#include "hip/hip_runtime.h"
#ifndef _META_CU_
#define _META_CU_

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <hiprand/hiprand_kernel.h>
#include <GPU_TS.cuh>
#include <CPU_TS.cuh>
#include <hiprand.h>
#include "mpi.h"
#include "data.h"

/******************************************************************************************************/
/***************************************USER DECLARATION***********************************************/
/******************************************************************************************************/

//********
//Activation
bool GPU;
bool CPU;

//********
//CPU Data
const int infinite = 2147483647;//max int
const int FALSE = 0;
const int TRUE = 1;

double  somme_sol = 0.0;
double  avg_time = 0.0;
double  cost_sol;

typedef int*  type_vector;
typedef int** type_matrix;

int max_global_iteration;
double  stop_condition;
int L;//limit of stagnacy

type_vector individual;           /* current solution (permutation) */
int fitness;                      /* current cost                   */
type_vector best_individual;      /* best solution                  */
int best_fitness;                 /* best cost                      */

int w;    /* history of search parameters       */
int step; /* step of the glover diversification */

bool Stangnancy;
int perturb; /* perturbation parameters */
int TS_iterations; /* number of Tabu Search iteration*/

clock_t iter_end,start,end;

//********
//GPU Data
int          *c_a;       /* Matrix of flows     */
int          *c_b;       /* Matrix of distances */
int          *g_delta;   /* Matrix of costs     */
int          *g_p;       /* current solution    */

/***************************************End User Declaration******************************************/

/***************************************************************************************************/
/***************************************USER FONCTION***********************************************/
/***************************************************************************************************/

/*************** L'Ecuyer random number generator ***************/
double rando()
 {
  static int x10 = 12345, x11 = 67890, x12 = 13579, /* initial value*/
             x20 = 24680, x21 = 98765, x22 = 43210; /* of seeds*/
  const int m = 2147483647; const int m2 = 2145483479;
  const int a12= 63308; const int q12=33921; const int r12=12979;
  const int a13=-183326; const int q13=11714; const int r13=2883;
  const int a21= 86098; const int q21=24919; const int r21= 7417;
  const int a23=-539608; const int q23= 3976; const int r23=2071;
  const double invm = 4.656612873077393e-10;
  int h, p12, p13, p21, p23;
  h = x10/q13; p13 = -a13*(x10-h*q13)-h*r13;
  h = x11/q12; p12 = a12*(x11-h*q12)-h*r12;
  if (p13 < 0) p13 = p13 + m; if (p12 < 0) p12 = p12 + m;
  x10 = x11; x11 = x12; x12 = p12-p13; if (x12 < 0) x12 = x12 + m;
  h = x20/q23; p23 = -a23*(x20-h*q23)-h*r23;
  h = x22/q21; p21 = a21*(x22-h*q21)-h*r21;
  if (p23 < 0) p23 = p23 + m2; if (p21 < 0) p21 = p21 + m2;
  x20 = x21; x21 = x22; x22 = p21-p23; if(x22 < 0) x22 = x22 + m2;
  if (x12 < x22) h = x12 - x22 + m; else h = x12 - x22;
  if (h == 0) return(1.0); else return(h*invm);
 }

/*********** return an integer between low and high *************/
int unif(int low, int high)
 {return low + (int)((double)(high - low + 1) * rando()) ;}

void transpose(int *a, int *b) {int temp = *a; *a = *b; *b = temp;}

int minim(int a, int b) {if (a < b) return(a); else return(b);}

double cube(double x) {return x*x*x;}


void generate_random_solution(int n, type_vector  p)
 {int i;
  for (i = 0; i < n;   i++) p[i] = i;
  for (i = 0; i < n-1; i++) transpose(&p[i], &p[unif(i, n-1)]);
 }

/***************************************End User Fonction******************************************/

__host__ void Meta_Init(int n,type_vector a,type_vector b)
{
  //********
  //Activation
  GPU = TRUE;
  CPU = FALSE;

  if(GPU) printf("GPGPU optimizer activated\n");
  if(CPU) printf("CPU optimizer activated\n");
  //********
  //GPU allocation and initialization
  printf("GPU init Data...\n");

  int dim;
  dim = n * n * sizeof(int);

  //Allocation for distance and flow matrices
  hipMalloc((void **)&c_a, dim);
  cutilCheckMsg("Meta_init a: hipMalloc() execution failed\n");
  hipMemset(c_a, 0, dim);
  cutilCheckMsg("Meta_init a: hipMemset() execution failed\n");
  hipMemcpy(c_a, a, (n * n * sizeof(int)), hipMemcpyHostToDevice);
  cutilCheckMsg("Meta_init a: hipMemcpy() execution failed\n");

  hipMalloc((void **)&c_b, dim);
  cutilCheckMsg("Meta_init b: hipMalloc() execution failed\n");
  hipMemset( c_b, 0, dim);
  cutilCheckMsg("Meta_init b: hipMemset() execution failed\n");
  hipMemcpy(c_b, b, (n * n * sizeof(int)), hipMemcpyHostToDevice);
  cutilCheckMsg("Meta_init b: hipMemcpy() execution failed\n");

  //Allocation for the delta matrix
  hipMalloc((void **)&g_delta, dim);
  cutilCheckMsg("Meta_init delta: hipMalloc() execution failed\n");
  hipMemset( g_delta, 0, dim);
  cutilCheckMsg("Meta_init delta: hipMemset() execution failed\n");

  dim = n * sizeof(int);

  //Allocation for the vector of solution p
  hipMalloc((void **)&g_p, dim);
  cutilCheckMsg("Meta_init p: hipMalloc() execution failed\n");
  hipMemset( g_p, 0, dim);
  cutilCheckMsg("Meta_init p: hipMemset() execution failed\n");

  //********
  //CPU allocation and initialization
  printf("CPU init Data...\n");

  TS_iterations = n*1000; /* SIZE MULTIPLY BY 1000*/
  max_global_iteration = 200;
  if(n <= 100) stop_condition = 3600;//1 h
  else stop_condition = 14400;//4 h

  cost_sol  = 0.0;

  individual = (int*)calloc(n, sizeof(int));
  best_individual = (int*)calloc(n, sizeof(int));

  best_fitness = infinite;

  L = 20;
  w = 0;
  step = 3;

  Stangnancy = FALSE;
  perturb    = n/4; /* 25% of the solution */

}


__host__ void Meta_Optimize(int n,int BKS,type_vector a,type_vector b)
{

  start = clock();

  int i, k,position,star;
  int global_iter = 0;

  generate_random_solution(n, individual);

  do{
	if(GPU)
	{
	   	tabu_search_parallel(n, a, b,                /* problem data         */
	            	 individual, &fitness,               /* tabu search results  */
	             	 8*n, n*n*5,                         /* parameters           */
	               TS_iterations,BKS,                   /* number of iterations */
			           c_a, c_b, g_delta, g_p);             /* GPU DATA*/
	}

	if(CPU)
	{
	   	tabu_search_seq(n, a, b,                    /* problem data         */
		     	 individual, &fitness,              /* tabu search results  */
		     	 8*n, n*n*5,                        /* parameters           */
		 	 TS_iterations,BKS);                /* number of iterations */
        }
        /* Amelioration */
	if(fitness < best_fitness){

		Stangnancy = FALSE;
		w=0;
		best_fitness = fitness;

        	for (k = 0; k < n; k = k+1) best_individual[k] = individual[k];

        }
	else w++;
	/*End Amelioration part */

	if (w == L){ /* stagnancy*/
		Stangnancy = TRUE;
	}

	if (Stangnancy == FALSE){
		/* re-construction of the solution using the Golver method */  /* Rule 1 */
		position = 0;
		for (star = step; star >= 1; star--){
			for (k = (star-1); k < n; k = k+step){
				individual[position] = best_individual[k];
				position++;
			}
		}
		if(step == (n-1)) step=3;//reinitialization
		else step++;
	}//End reconstruction

	else {
		Stangnancy = FALSE;
		w=0;
		for (i = 0; i < n-1; i++) transpose(&individual[i], &individual[unif(i, n-1)]);/* Rule 2 */
	}//End re-localization

    	if(w == (L/2)){
		for (i = perturb; i < n-1; i++) transpose(&individual[i], &individual[unif(i, n-1)]); /* Rule 3 */
	}

	iter_end = clock();

        global_iter++;

   }while( global_iter < max_global_iteration /*&& best_fitness > BKS*/ );
   //}while( ((double)(iter_end-start)/CLOCKS_PER_SEC) < stop_condition && best_fitness > BKS );

   end = clock();

   cost_sol = best_fitness;

}

// Display and write results
__host__ void Meta_Display_results(FILE* fres, int n, int BKS){

   printf(" Best solution value found by GPU_HITS: %f \n", cost_sol);
   printf(" Best deviation found by GPU_HITS: %f \n", 100*(cost_sol - BKS)/BKS);
   for (int i = 0; i < n; i = i+1) printf("%d- ", (best_individual[i]+1));/* +1 pour l'affichage */
   printf("\n");
   printf(" Execution time = [%.3lf] second\n", (double)(end-start)/CLOCKS_PER_SEC);

   fprintf(fres,"%.3lf\t",(double)(end-start)/CLOCKS_PER_SEC);
   //fprintf(fres,"%f\n",100*(cost_sol - BKS)/BKS);
    fprintf(fres,"%f\n",cost_sol);

    somme_sol += best_fitness;
    avg_time += (double)(end-start)/CLOCKS_PER_SEC;

}


// Frees resources
__host__ void Meta_Free(){
	//**********************
	//  ARRAYS DE-ALLOCATION
	//**********************

	hipFree(c_a);
	cutilCheckMsg("h_HITS_Free a: hipFree() execution failed\n");
	hipFree(c_b);
	cutilCheckMsg("h_HITS_Free b: hipFree() execution failed\n");
	hipFree(g_delta);
	cutilCheckMsg("h_HITS_Free delta: hipFree() execution failed\n");
	hipFree(g_p);
	cutilCheckMsg("h_HITS_Free P: hipFree() execution failed\n");

	free(individual);
	free(best_individual);
}

// Display and write global results
__host__ void Meta_Display_trials_results(FILE* fres, int max_runs, int BKS){

   printf("****Recap****\n");
   printf("Average cost: %f, average dev: %f\n", somme_sol/max_runs, 100*(somme_sol/max_runs - BKS)/BKS);
   printf("Average time: %.3lf\n", avg_time/max_runs);

   fprintf(fres,"***AVG*** \n");
   fprintf(fres,"%.3lf\t",avg_time/max_runs);
   fprintf(fres,"%f\n",100*(somme_sol/max_runs - BKS)/BKS);
   fprintf(fres,"%f\n",somme_sol/max_runs);

   somme_sol = 0.0;
   avg_time  = 0.0;

}

#endif
