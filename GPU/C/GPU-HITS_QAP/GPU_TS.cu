#include "hip/hip_runtime.h"
#ifndef _GPU_TS_CU_
#define _GPU_TS_CU_

#include <GPU_Evaluation_QAP.cuh>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <hiprand/hiprand_kernel.h>

typedef int*  type_vector;
typedef int** type_matrix;

extern const int infinite;
extern const int FALSE;
extern const int TRUE;

double rando();

int unif(int low, int high);

void transpose(int *a, int *b);

int minim(int a, int b);

double cube(double x);

static __global__ void g_update_p(int* g_p, int i_retained, int j_retained){

	int temp = g_p[i_retained];
	g_p[i_retained] = g_p[j_retained];
	g_p[j_retained] = temp;

}


__host__ void tabu_search_parallel(int n,         /* problem size */
                 type_vector a,         	  /* flows matrix */
                 type_vector b,        	 	  /* distance matrix */
                 type_vector best_sol,  	  /* best solution found */
                 int *best_cost,        	  /* cost of best solution */
                 int tabu_duration,    	          /* parameter 1 (< n^2/2) */
                 int aspiration,        	  /* parameter 2 (> n^2/2)*/
                 int nr_iterations,int BKS,       /* number of iterations */ 
           	 int *c_a,int* c_b, int* g_delta, int* g_p)
 
{
	  type_vector p;                        /* current solution */
	  type_vector delta;                    /* store move costs */
	  type_matrix tabu_list;                /* tabu status */
	  int current_iteration;                /* current iteration */
	  int current_cost;                     /* current sol. value */
	  int i, j, k, i_retained, j_retained;  /* indices */
	  int min_delta;                        /* retained move cost */
	  int autorized;                        /* move not tabu? */
	  int aspired;                          /* move forced? */
	  int already_aspired;                  /* in case many moves forced */

	  /***************** dynamic memory allocation *******************/
	  p = (int*)calloc(n, sizeof(int));
	  delta = (int*)calloc((n*n),sizeof(int));
	  tabu_list = (int**)calloc(n,sizeof(int*));
	  for (i = 0; i < n; i = i+1) tabu_list[i] = (int*)calloc(n, sizeof(int));


	  /************** current solution initialization ****************/
	  for (i = 0; i < n; i = i + 1) p[i] = best_sol[i];

	  /********** initialization of current solution value ***********/
	  /**************** and matrix of cost of moves  *****************/
	  current_cost = 0;
	  for (i = 0; i < n; i = i + 1) for (j = 0; j < n; j = j + 1)
	   {
	    current_cost = current_cost + a[i*n+j] * b[p[i]*n+p[j]];
	   };

	  //hipMemcpy(g_delta, delta, (n * n * sizeof(int)), hipMemcpyHostToDevice);
	  //cutilCheckMsg("tabu cudaITSSD delta: hipMemcpy() execution failed\n");
	  hipMemcpy(g_p, p, (n * sizeof(int)), hipMemcpyHostToDevice);
	  cutilCheckMsg("tabu search delta: hipMemcpy() execution failed\n");

	  h_compute_delta(n, g_p, g_delta, c_a, c_b);

	   hipMemcpy(delta, g_delta, (n * n * sizeof(int)), hipMemcpyDeviceToHost);
	   cutilCheckMsg("tabu search DeviceToHost delta: hipMemcpy() execution failed\n");
	  
	  *best_cost = current_cost;

	  /****************** tabu list initialization *******************/
	  for (i = 0; i < n; i = i + 1) for (j = 0; j < n; j = j+1)
	    tabu_list[i][j] = -(n*i + j);

	  /******************** main tabu search loop ********************/
	  for (current_iteration = 1; current_iteration <= nr_iterations && *best_cost > BKS; current_iteration++ )
	   {/** find best move (i_retained, j_retained) **/
	    i_retained = infinite;       /* in case all moves are tabu */
	    j_retained = infinite;
	    min_delta = infinite;
	    already_aspired = FALSE;
	    
	    for (i = 0; i < n-1; i = i + 1) 
	      for (j = i+1; j < n; j = j+1)
	       {autorized = (tabu_list[i][p[j]] < current_iteration) || (tabu_list[j][p[i]] < current_iteration);
		aspired = (tabu_list[i][p[j]] < current_iteration-aspiration)||(tabu_list[j][p[i]] < current_iteration-aspiration)||
		          (current_cost + delta[i*n+j] < *best_cost);                

		if ((aspired && !already_aspired) || /* first move aspired */
		   (aspired && already_aspired &&    /* many move aspired  */
		    (delta[i*n+j] < min_delta)) ||   /* => take best one   */
		   (!aspired && !already_aspired &&  /* no move aspired yet*/
		    (delta[i*n+j] < min_delta) && autorized))
		  {i_retained = i; j_retained = j;
		   min_delta = delta[i*n+j];
		   if (aspired) {already_aspired = TRUE;};};};

	    if (i_retained == infinite) printf("All moves are tabu! \n"); 
	    else 
	     {/** transpose elements in pos. i_retained and j_retained **/
	      transpose(&p[i_retained], &p[j_retained]);
	      /* update solution value*/
	      current_cost = current_cost + delta[i_retained*n+j_retained];
	      /* forbid reverse move for a random number of iterations*/
	      tabu_list[i_retained][p[j_retained]] = current_iteration + (int)(cube(rando())*tabu_duration);
	      tabu_list[j_retained][p[i_retained]] = current_iteration + (int)(cube(rando())*tabu_duration);

	      /* best solution improved ?*/
	      if (current_cost < *best_cost)
	       {*best_cost = current_cost;
		for (k = 0; k < n; k = k+1) best_sol[k] = p[k];
		//printf("Solution of value: %d found at iter. %d\n", current_cost, current_iteration);
	       };  
	   
		g_update_p<<<1, 1>>>(g_p, i_retained, j_retained);
		cutilCheckMsg("tabu search: g_update_p() execution failed\n");


	        h_update_delta(n, g_p, g_delta, c_a, c_b, i_retained, j_retained);

	        hipMemcpy(delta, g_delta, (n * n * sizeof(int)), hipMemcpyDeviceToHost);
	        cutilCheckMsg("TS DeviceToHost delta: hipMemcpy() execution failed\n");
	     
	     };
	   }; 
	  /* free memory*/
	  free(p);
	  free(delta);
	  for (i=0; i < n; i = i+1) free(tabu_list[i]); free(tabu_list);
} /* parallel tabu*/

#endif


